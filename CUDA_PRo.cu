
#include <hip/hip_runtime.h>
#include<stdio.h>
#include<fstream>
//////////////////////////////////////////////////////////////
__global__ void Conv3x3_2d(const double *g_input,const double *g_weight3x3_2d,double *g_output3x3_2d) { 
    __shared__ double s_weight3x3_2d[9];    //using shared mem for only weight
    for(int i=0;i<9;i++){
        s_weight3x3_2d[i] = g_weight3x3_2d[i];
    }
    double sum=0;
    int x= blockIdx.x*blockDim.x + threadIdx.x;
    int y = blockIdx.y * blockDim.y + threadIdx.y;
    int i = y*224+x; //for c[i] = sum;

    for(int z=0;z<300;z++) {
      sum=0;    //reset sum 
      for(int m=0;m<3;m++){   //3x3
          for(int n=0;n<3;n++){
              sum += g_input[z*226*226 + (y+m)*226+(x+n)] * s_weight3x3_2d[m*3 +n];   //convolution
          }
      }
      g_output3x3_2d[i+ z*(224)*(224)] =sum;    //return sum result to output
    }
}


////////////////////////////////////////////////////////////
__global__ void Conv1x1_3d(const double *g_input,const double *g_weight1x1_3d,double *g_output1x1_3d){
    __shared__ double s_weight1x1_3d[1*1*300];    //using shared mem for only weight
    for(int i=0;i<1*1*300;i+=2){
        s_weight1x1_3d[i]=g_weight1x1_3d[i];
        s_weight1x1_3d[i+1]=g_weight1x1_3d[i+1];
    }
     double sum=0;
    int x= blockIdx.x*blockDim.x + threadIdx.x;
    int y = blockIdx.y * blockDim.y + threadIdx.y;
    int z = threadIdx.z;
    int i= z*224*224+y*224+x;

    for(int a=0;a<900;a++) {    //output channel
      sum=0;    //reset for next convolution
      for(int v=0;v<300;v++){
        for(int m=0;m<1;m++){   //1x1
         for(int n=0;n<1;n++){
             if(a %2 ==0){
              sum += g_input[(z+v)*226*226+(y+m+1)*226+(x+n+1)] *1; // s_weight1x1_3d[v+m+n];   //convolution
             }
             else{
              sum += g_input[(z+v)*226*226+(y+m+1)*226+(x+n+1)] * -1; // -s_weight1x1_3d[v+m+n];   //convolution
             }
         }
       }
      }
      g_output1x1_3d[i+ a*224*224] =sum;   //return sum result to output
    }
}

/////////////////////////////////////////////////////////////
__global__ void Conv3x3_3d(const double *g_input,const double *g_weight3x3_3d,double *g_output3x3_3d){
      __shared__ double s_weight3x3_3d[3*3*300];    //use shared mem
    for(int i=0;i<3*3*300;i+=2){        //use loop unrolling 
        s_weight3x3_3d[i]=g_weight3x3_3d[i];
        s_weight3x3_3d[i+1]=g_weight3x3_3d[i+1];
    }
     double sum=0;
    int x= blockIdx.x*blockDim.x + threadIdx.x;
    int y = blockIdx.y * blockDim.y + threadIdx.y;
    int z = threadIdx.z;
    int i= z*224*224+y*224+x;

    for(int a=0;a<900;a++) {  //output channel
      sum=0;    //reset for next convolution
      for(int v=0;v<300;v++){
        for(int m=0;m<3;m++){ //3x3
         for(int n=0;n<3;n++){   
             sum += g_input[(z+v)*226*226 + (y+m)*226+(x+n)] * s_weight3x3_3d[v*3*3 + m*3 +n]; //convolution
         }
       }
      }
      g_output3x3_3d[i+ a*224*224] =sum;   //return sum result to output
    }
}

//////////////////////////////////////////////////////////////
void input(double * input)
{
    for(int z=0;z<300;z++)
        for(int i=0;i<226;i++)
            for(int j=0;j<226;j++)
            {
                if(i == 0 || j == 0 || j == 225 || i == 225)
                    input[z*226*226+i*226+j] = 0;
                else
                    input[z*226*226+i*226+j] = 1;
            }
}

void Init_weight3x3_3d(double * weight)
{
    for(int k=0;k<900;k++)
        for(int z=0;z<300;z++)
            for(int i=0;i<3;i++)
                for(int j=0;j<3;j++){
                    if((i*3 + j) % 2 == 0)
                        weight[k*3*3*300 + z*3*3+i*3+j] = 1;
                    else
                        weight[k*3*3*300 + z*3*3+i*3+j] = -1;
                }
}

void Init_weight3x3_2d(double * weight)
{
    for(int i=0;i<9;i++)
        weight[i] = 1;
}

void Init_weight1x1_3d(double * weight)
{
    for(int k = 0;k<900;k++)
        for(int z=0;z<300;z++){
            if(k % 2 == 0)
                weight[z] = 1;	//짝수
            else
                weight[z] = -1;	//홀수
        }
}

void save_file(double * output3x3_3d,double * output1x1_3d,double * output3x3_2d)
{
    FILE * fp = fopen("3d_Result3x3.txt","w");
    FILE * fp2 = fopen("3d_Result1x1.txt","w");
    FILE * fp3 = fopen("2d_Result3x3.txt","w");
    for(int i=0;i<224*224*900;i++)
    {
        fprintf(fp,"%f\n",output3x3_3d[i]);
        fprintf(fp2,"%f\n",output1x1_3d[i]);
    }
    for(int i=0;i<224*224*300;i++)
        fprintf(fp3,"%f\n",output3x3_2d[i]);

    fclose(fp3);
    fclose(fp2);
    fclose(fp);
}
int main(){
    
    double * input_3d = (double*)malloc(sizeof(double)*226*226*300);    //input Feature Map
    
    double * output3x3_2d = (double*)malloc(sizeof(double)*224*224*300);//Output Feature Map
    double * output1x1_3d = (double*)malloc(sizeof(double)*224*224*900);
    double * output3x3_3d = (double*)malloc(sizeof(double)*224*224*900);

    double * weight3x3_2d = (double*)malloc(sizeof(double)*3*3);		//Weight
    double * weight1x1_3d = (double*)malloc(sizeof(double)*1*1*300*900);
    double * weight3x3_3d = (double*)malloc(sizeof(double)*3*3*300*900);
    
    double * g_input, * g_output3x3_3d, * g_output1x1_3d, * g_output3x3_2d, * g_weight3x3_3d, * g_weight1x1_3d, * g_weight3x3_2d;
    
    hipEvent_t start, stop3x3_3d, stop1x1_3d, stop3x3_2d;
    
    hipEventCreate(&start);
    hipEventCreate(&stop3x3_3d);
    hipEventCreate(&stop1x1_3d);
    hipEventCreate(&stop3x3_2d);
    //Initialization Input Feature Map & Weight
    input(input_3d);
    Init_weight3x3_3d(weight3x3_3d);
    Init_weight1x1_3d(weight1x1_3d);
    Init_weight3x3_2d(weight3x3_2d);
    
    hipMalloc((void**)&g_input,sizeof(double)*226*226*300);
    hipMalloc((void**)&g_output3x3_3d,sizeof(double)*224*224*900);
    hipMalloc((void**)&g_output1x1_3d,sizeof(double)*224*224*900);
    hipMalloc((void**)&g_output3x3_2d,sizeof(double)*224*224*300);
    hipMalloc((void**)&g_weight3x3_3d,sizeof(double)*3*3*300*900);
    hipMalloc((void**)&g_weight1x1_3d,sizeof(double)*1*1*300*900);
    hipMalloc((void**)&g_weight3x3_2d,sizeof(double)*3*3);
    
    hipMemcpy(g_input,input_3d,sizeof(double)*226*226*300,hipMemcpyHostToDevice);
    hipMemcpy(g_weight3x3_3d,weight3x3_3d,sizeof(double)*3*3*300*900,hipMemcpyHostToDevice);
    hipMemcpy(g_weight1x1_3d,weight1x1_3d,sizeof(double)*1*1*300*900,hipMemcpyHostToDevice);
    hipMemcpy(g_weight3x3_2d,weight3x3_2d,sizeof(double)*3*3,hipMemcpyHostToDevice);

    /*
    Project
    Block 및 Grid 선언 자유, 주어진 3개의 Kernel Conv3x3_3d, Conv1x1_3d, Conv3x3_2d를 구현(Kernel명 및 Argument 유지)
    가능한 빠른 Performance를 가지는 Kernel을 구현할 것

    결과는 Text File을 통해서 확인, cudaEvent 관련 코드는 성능 측정을 위한 코드이니 수정하지 말것
	
	Kernel 별 배점
	Conv3x3_2d = 20%
	Conv1x1_3d = 35%
	Conv3x3_3d = 45%
    */
////////////////////////////////////////아래의 3개 Kernel을 구현 ///////////////////////////////////
    dim3 DimGrid(7,7);    //7by7 for 7*32 = 224 
    dim3 DimBlock(32,32,1); //32*32 = 1024 use max thread
    hipEventRecord(start);
    Conv3x3_2d<<<DimGrid,DimBlock>>>(g_input,g_weight3x3_2d,g_output3x3_2d);
    hipEventRecord(stop3x3_2d);
	
    Conv1x1_3d<<<DimGrid,DimBlock>>>(g_input,g_weight1x1_3d,g_output1x1_3d);
    hipEventRecord(stop1x1_3d);
	
    Conv3x3_3d<<<DimGrid,DimBlock>>>(g_input,g_weight3x3_3d,g_output3x3_3d); 
    hipEventRecord(stop3x3_3d);

    hipEventSynchronize(stop3x3_3d);    
/////////////////////////////////////////////////////////////////////////////////////////////////

    float milliseconds[3]={0};
    hipEventElapsedTime(&milliseconds[0],start,stop3x3_2d);
    hipEventElapsedTime(&milliseconds[1],stop3x3_2d,stop1x1_3d);
    hipEventElapsedTime(&milliseconds[2],stop1x1_3d,stop3x3_3d);
    printf("Execution Time \n Convolution3x3_2d : %f\n Convolution1x1_3d : %f\n Convolution3x3_3d : %f\n",milliseconds[0],milliseconds[1],milliseconds[2]);
    
    hipMemcpy(output3x3_3d,g_output3x3_3d,sizeof(double)*224*224*900,hipMemcpyDeviceToHost);
    hipMemcpy(output1x1_3d,g_output1x1_3d,sizeof(double)*224*224*900,hipMemcpyDeviceToHost);
    hipMemcpy(output3x3_2d,g_output3x3_2d,sizeof(double)*224*224*300,hipMemcpyDeviceToHost);
  
    save_file(output3x3_3d,output1x1_3d,output3x3_2d);
    hipFree(g_input);
    hipFree(g_weight3x3_3d);
    hipFree(g_weight3x3_2d);
    hipFree(g_weight1x1_3d);
    hipFree(g_output3x3_3d);
    hipFree(g_output3x3_2d);
    hipFree(g_output1x1_3d);

    free(output3x3_3d);
    free(output1x1_3d);
    free(output3x3_2d);
    free(input_3d);
    free(weight3x3_3d);
    free(weight1x1_3d);
    free(weight3x3_2d);
}
